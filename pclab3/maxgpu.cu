#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define TPB 1024 //elements per thread
#define BN 64    //block number

/*function declarations*/
int getmax(int *, int);
__global__ void kernel_getmax(int *, int, int);
__device__ void thread_getmax(int *, int *, int, int);

//the sequential version of getmax
int getmax(int num[], int size){
    int i;
    int max=num[0];
    for(i=1;i<size;i++){
	if(num[i]>max){
	    max=num[i];
	}
    }
    return max;
}

/*kernel called by the host to getmax. 
  The high level idea is that each thread first find the max in its share of TPB elements. 
  Then each block use the reduction tree algorithm to find the max in the block. 
  After the two steps above, the max can be found among the first BN elements of the array num[]. 
  We move them back to CPU and use the sequential version to find the max among the last BN elements.*/
__global__ void kernel_getmax(int num[], int size, int workload)
{
    //first, we ask each thread to find the max in its assigned EPT random numbers
    __shared__ int max_each_thread[TPB]; 
    thread_getmax(num, max_each_thread, size, workload);
    __syncthreads();
    
    //next, we find the max in a block. note that the same tree algorithm for parallel summation applies to max as well
    int thread_id=threadIdx.x;
    //loop unrolling for efficiency
    if(thread_id<512){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+512]){
            max_each_thread[thread_id]=max_each_thread[thread_id+512];
        }
    }
    __syncthreads();
    if(thread_id<256){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+256]){
            max_each_thread[thread_id]=max_each_thread[thread_id+256];
        }
    }
    __syncthreads();
    if(thread_id<128){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+128]){
            max_each_thread[thread_id]=max_each_thread[thread_id+128];
        }
    }
    __syncthreads();
    if(thread_id<64){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+64]){
            max_each_thread[thread_id]=max_each_thread[thread_id+64];
        }
    }
    __syncthreads();
    if(thread_id<32){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+32]){
            max_each_thread[thread_id]=max_each_thread[thread_id+32];
        }
    }
    __syncthreads();
    if(thread_id<16){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+16]){
            max_each_thread[thread_id]=max_each_thread[thread_id+16];
        }
    }
    __syncthreads();
    if(thread_id<8){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+8]){
            max_each_thread[thread_id]=max_each_thread[thread_id+8];
        }
    }
    __syncthreads();    
    if(thread_id<4){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+4]){
            max_each_thread[thread_id]=max_each_thread[thread_id+4];
        }
    }
    __syncthreads();
    if(thread_id<2){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+2]){
            max_each_thread[thread_id]=max_each_thread[thread_id+2];
        }
    }
    __syncthreads();
    if(thread_id<1){
        if(max_each_thread[thread_id]<max_each_thread[thread_id+1]){
            max_each_thread[thread_id]=max_each_thread[thread_id+1];
        }
    }
    __syncthreads();

    //we put the max of the i-th block at num[i]
    if(thread_id==0){
	num[blockIdx.x]=max_each_thread[0];
    }
}

/*The function called by the kernel. The sequential getmax version for each thread*/
__device__ void thread_getmax(int num[], int max_each_thread[], int size, int workload){
    int max=0;
    int index=workload*(blockIdx.x*blockDim.x+threadIdx.x);
    int i;
    for(i=index;(i<size)&&(i<index+workload);i++){
	    if(max<num[i]){
		max=num[i];
	    }
    }

    //store the max of this thread in the corresponding position of the shared array
    max_each_thread[threadIdx.x]=max;
}


int main(int argc, char *argv[])
{
    int size = 0;  // The size of the array
    int i;  // loop index
    int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxgpu num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (int *)malloc(size * sizeof(int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    

    /*todo: 1)allocate memory and copy numbers from host to device 
	    2)invoke kernels to deal with the array
	    3)copy numbers from device to host and free memory*/
    
    int workload=ceil((double)size/(TPB*BN));
    //step1:memory setup
    int *gpu_numbers;
    hipError_t err;
    err=hipMalloc((void**)&gpu_numbers, sizeof(int)*size);
    //sometimes when x=100,000,000, we might fail to allocate/transfer memory
    if(err!=hipSuccess){
	printf("Cannot allocate memory for the initial random array\n");
    }
    err=hipMemcpy(gpu_numbers, numbers, sizeof(int)*size, hipMemcpyHostToDevice);
    if(err!=hipSuccess){
	printf("cannot pass the random array from cpu to gpu\n");
    }
    
    //step2:invoke kernal
    kernel_getmax<<<BN, TPB>>>(gpu_numbers, size, workload);
    
    //step3:copy the max from device & housekeeping(free the allocated pointers)
    hipMemcpy(numbers,gpu_numbers, sizeof(int)*BN, hipMemcpyDeviceToHost);
    printf(" The maximum number calculated from GPU is: %d\n", getmax(numbers, BN));
    hipFree(gpu_numbers);
    free(numbers);
    exit(0);
}


